#include "hip/hip_runtime.h"
/*
Benjamin A. Slack
CS5260
CUDA Addition of 2D Matricies
11.30.17
*/

// Developing this on Windows so that I have access to 
// my GPU for testing. However, time.h and sys/time.h
// are not available in Windows for reasons of M$ infinite
// wisdom. Therefore using this snippet from Stack Overflow
// to make a platform indepedent timing function. Should 
// make it so I can compile on windows or Thor as needed.
//
// URL:
// https://stackoverflow.com/questions/17432502/how-can-i-measure-cpu-time-and-wall-clock-time-on-both-linux-windows

//  Windows
#ifdef _WIN32
#include <Windows.h>
double get_wall_time() {
	LARGE_INTEGER time, freq;
	if (!QueryPerformanceFrequency(&freq)) {
		//  Handle error
		return 0;
	}
	if (!QueryPerformanceCounter(&time)) {
		//  Handle error
		return 0;
	}
	return (double)time.QuadPart / freq.QuadPart;
}
double get_cpu_time() {
	FILETIME a, b, c, d;
	if (GetProcessTimes(GetCurrentProcess(), &a, &b, &c, &d) != 0) {
		//  Returns total user time.
		//  Can be tweaked to include kernel times as well.
		return
			(double)(d.dwLowDateTime |
			((unsigned long long)d.dwHighDateTime << 32)) * 0.0000001;
	}
	else {
		//  Handle error
		return 0;
	}
}

//  Posix/Linux
#else
#include <time.h>
#include <sys/time.h>
double get_wall_time() {
	struct timeval time;
	if (gettimeofday(&time, NULL)) {
		//  Handle error
		return 0;
	}
	return (double)time.tv_sec + (double)time.tv_usec * .000001;
}
double get_cpu_time() {
	return (double)clock() / CLOCKS_PER_SEC;
}
#endif

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include "hip/driver_types.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <args.h>
//#include <errno.h>
#include <stdbool.h>
//#include <book.h>
//#include <math.h>

#define TOK_MATRIXHEIGHT "-m"
#define TOK_MATRIXWIDTH "-n"
#define TOK_ONCPU "-c"
#define TOK_BLOCKSHEIGHT "-y"
#define TOK_BLOCKSWIDTH "-x"
#define TOK_THREADSX "-tx"
#define TOK_THREADSY "-ty"
#define DEFAULT_BLOCKSDIM 16
#define DEFAULT_THREADSX 8
#define DEFAULT_THREADSY 4
#define DEFAULT_M 128
#define DEFAULT_N 128
#define DEFAULT_GPU true;

enum Location {HOST, DEVICE};

typedef struct matrix {
	//Location loc;
	int m;
	int n;
	float *contents;
}matrix_t;

#define calc_matrix_offset(_PTR, _M, _N) (_M + _N*_PTR->m)

// Borrowed this macro from Stack Overflow for generating
// custom cuda error messages.
// https://stackoverflow.com/questions/16282136/is-there-a-cuda-equivalent-of-perror
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

matrix_t *alloc_matrix(Location, int, int);

//__host__ __device__ int calc_matrix_offset(matrix_t *, int, int);

__host__ __device__ float get_matrix_item(matrix_t *, int, int);

__host__ __device__ void set_matrix_item(matrix_t *, int, int, float);

void free_matrix(matrix_t *, Location);

void copy_matrix(matrix_t *, matrix_t *, Location);

__host__ __device__ void init_A(matrix_t *, int, int);

__host__ __device__ void init_B(matrix_t *, int, int);

__global__ void setupA(matrix_t *);

__global__ void setupB(matrix_t *);

__global__ void add(matrix_t *, matrix_t *, matrix_t *);

int main(int argc, char **argv) {
	//init
	int m = DEFAULT_M;
	int n = DEFAULT_N;
	int x = DEFAULT_BLOCKSDIM;
	int y = DEFAULT_BLOCKSDIM;
	int tx = DEFAULT_THREADSX;
	int ty = DEFAULT_THREADSY;
	bool run_on_gpu = DEFAULT_GPU;
	matrix_t *a = NULL;
	matrix_t *b = NULL;
	matrix_t *c = NULL;
	matrix_t *dev_a = NULL;
	matrix_t *dev_b = NULL;
	matrix_t *dev_c = NULL;

	//parse command line
	for (int i = 1; i < argc; i++) {
		if (strcmp(argv[i], TOK_MATRIXHEIGHT) == 0) {
			m = atoi(argv[i + 1]);
		}
		if (strcmp(argv[i], TOK_MATRIXWIDTH) == 0) {
			n = atoi(argv[i + 1]);
		}
		if (strcmp(argv[i], TOK_ONCPU) == 0) {
			run_on_gpu = false;
		}
		if (strcmp(argv[i], TOK_BLOCKSWIDTH) == 0) {
			x = atoi(argv[i + 1]);
		}
		if (strcmp(argv[i], TOK_BLOCKSHEIGHT) == 0) {
			y = atoi(argv[i + 1]);
		}
		if (strcmp(argv[i], TOK_THREADSX) == 0) {
			tx = atoi(argv[i + 1]);
		}
		if (strcmp(argv[i], TOK_THREADSY) == 0) {
			ty = atoi(argv[i + 1]);
		}
	}
	dim3 grid = { (unsigned int)x, (unsigned int)y };
	dim3 threads = { (unsigned int)tx, (unsigned int)ty };

	//create matricies
	if (run_on_gpu) {
		// might as well init matricies on the GPU
		dev_a = alloc_matrix(DEVICE, m, n);
		dev_b = alloc_matrix(DEVICE, m, n);
		dev_c = alloc_matrix(DEVICE, m, n);
		setupA KERNEL_ARGS2(grid, threads) (dev_a);
		cudaCheckErrors("main: setupA");
		setupB KERNEL_ARGS2(grid, threads) (dev_b);
		cudaCheckErrors("main: setupB");
	}

	// we need matricies on the host regardless
	a = alloc_matrix(HOST, m, n);
	b = alloc_matrix(HOST, m, n);
	c = alloc_matrix(HOST, m, n);
	// init the host matricies
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			init_A(a, i, j);
			init_B(b, i, j);
		}
	}
	
	//if we're running on the GPU, do the additions
	double gpu_start = 0.0;
	double gpu_end = 0.0;
	
	if (run_on_gpu) {
		//start timing
		gpu_start = get_wall_time();
		//add matricies
		add KERNEL_ARGS2(grid, threads) (dev_a, dev_b, dev_c);
		cudaCheckErrors("main: add");
		//get matricies from GPU
		//copy_matrix(a, dev_a, DEVICE);
		//copy_matrix(b, dev_b, DEVICE);
		copy_matrix(c, dev_c, DEVICE);
		//stop timing
		gpu_end = get_wall_time();
	}

	//need to make a matching matrix on CPU for checking
	matrix_t *check = alloc_matrix(HOST, m, n);

	// will need timing data for cpu, regardless
	//start timing
	double cpu_start = 0.0;
	double cpu_end = 0.0;
	cpu_start = get_wall_time();
	
	//add matricies on CPU
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			float temp = get_matrix_item(a, i, j) + get_matrix_item(b, i, j);
			set_matrix_item(check, i, j, temp);
		}
	}
	//stop timing
	cpu_end = get_wall_time();

	//if we're running on the GPU we check the returns
	if (run_on_gpu) {
		//check matrix addition
		bool checked = true;
		for (int i = 0; i < m; i++) {
			for (int j = 0; j < n; j++) {
				checked &= (get_matrix_item(c, i, j) == get_matrix_item(check, i, j));
			}
		}
		if (checked) {
			printf("We did it!\n\r");
		}
		else {
			printf("Something borked!\n\r");
		}
	}

	//if we're running on the gpu, output the elapsed gpu and cpu time
	if (run_on_gpu) {
		printf("m: %d, n: %d, grid: %d x %d, threads: %d, elapsed gpu time: %f\n\r", \
			m, n, x, y, (int)(tx * ty), gpu_end - gpu_start);
	}
	printf("m: %d, n: %d, grid: %d x %d, threads: %d, elapsed cpu time: %f\n\r", \
		m, n, x, y, (int)(tx * ty), cpu_end - cpu_start);

	free_matrix(a, HOST);
	free_matrix(b, HOST);
	free_matrix(c, HOST);
	free_matrix(dev_a, DEVICE);
	free_matrix(dev_b, DEVICE);
	free_matrix(dev_c, DEVICE);
	free_matrix(check, HOST);

	exit(0);
}

matrix_t *alloc_matrix(Location loc, int m, int n) {
	matrix_t *ret_ptr = NULL;
	if (loc == HOST) {
		ret_ptr = (matrix_t *)calloc(1, sizeof(matrix_t));
		//ret_ptr->loc = loc;
		ret_ptr->m = m;
		ret_ptr->n = n;
		ret_ptr->contents = (float *)calloc(m*n, sizeof(float));
	}
	else {
		matrix_t *temp = NULL;
		temp = (matrix_t *)calloc(1, sizeof(matrix_t));
		//temp->loc = loc;
		temp->m = m;
		temp->n = n;
		hipMalloc((void **)&temp->contents, sizeof(float)*m*n);
		cudaCheckErrors("alloc_matrix: contents mem");
		hipMalloc((void **)&ret_ptr, sizeof(matrix_t));
		cudaCheckErrors("alloc_matrix: matrix mem");
		hipMemcpy((void *)ret_ptr, (void *)temp, sizeof(matrix_t), hipMemcpyHostToDevice);
		cudaCheckErrors("alloc_matrix: matrix copy");
		free(temp);
	}
	return ret_ptr;
}

/*
__host__ __device__ int calc_matrix_offset(matrix_t *mat, int m, int n)
{
	return m + n*(mat->m);
}
*/

__host__ __device__ float get_matrix_item(matrix_t * mat, int  m, int n)
{
	return mat->contents[calc_matrix_offset(mat, m, n)];
}

__host__ __device__ void set_matrix_item(matrix_t * mat, int m, int n, float f)
{
	mat->contents[calc_matrix_offset(mat, m, n)] = f;
}

void free_matrix(matrix_t *mat, Location loc)
{
	if (loc == HOST){
		free(mat->contents);
		free(mat);
	}
	else {
		matrix_t *temp = NULL;
		temp = (matrix_t *)calloc(1, sizeof(matrix_t));
		hipMemcpy((void *)temp, (void *)mat, sizeof(matrix_t), hipMemcpyDeviceToHost);
		cudaCheckErrors("free_matrix: matrix copy");
		hipFree((void *)temp->contents);
		cudaCheckErrors("free_matrix: free contents");
		hipFree((void *)mat);
		cudaCheckErrors("free_matrix: matrix free");
		free(temp);
	}
}

void copy_matrix(matrix_t *dest, matrix_t *source, Location source_loc)
{
	if (source_loc == HOST) {
		// ok, dest already exists, so to get access to its contents ptr
		// we need to bring it back to the host
		matrix_t *temp_mat = (matrix_t *)calloc(1, sizeof(matrix_t));
		hipMemcpy((void *)temp_mat, (void *)dest, sizeof(matrix_t), hipMemcpyDeviceToHost);
		cudaCheckErrors("copy_matrix: temp_mat back copy");
		float *dev_mat_contents = temp_mat->contents;
		// now that we have the device pointer for its contents, we can copy
		// the contents from the source to the device
		hipMemcpy((void *)dev_mat_contents, (void *)source->contents, \
			sizeof(float)*source->m*source->n, hipMemcpyHostToDevice);
		cudaCheckErrors("copy_matrix: copy source contents to dest");
		// likely unnecessary, but just to make sure set the values from the source
		temp_mat->m = source->m;
		temp_mat->n = source->n;
		// ok now we can put it back, not, if we assume that m and n don't change
		// there's no reason to do this last copy
		hipMemcpy((void *)dest, (void *)temp_mat, sizeof(matrix_t), hipMemcpyHostToDevice);
		cudaCheckErrors("copy_matrix: update dest mat from temp");
		free(temp_mat);
	}
	else {
		//ok, dealing with a device matrix
		// we need to bring back the mat struct
		matrix_t *temp_mat = (matrix_t *)calloc(1, sizeof(matrix_t));
		hipMemcpy((void *)temp_mat, (void *)source, sizeof(matrix_t), hipMemcpyDeviceToHost);
		cudaCheckErrors("copy_matrix: get temp from source");
		// then copy the contents
		hipMemcpy((void *)dest->contents, (void *)temp_mat->contents, \
			sizeof(float)*temp_mat->m*temp_mat->n, hipMemcpyDeviceToHost);
		cudaCheckErrors("copy_matrix: copy contents from temp to dest");
		// and set the variables to be safe
		dest->m = temp_mat->m;
		dest->n = temp_mat->n;
		free(temp_mat);
	}
}

__host__ __device__ void init_A(matrix_t *mat, int i, int j)
{
	mat->contents[calc_matrix_offset(mat, i, j)] = 2.0 * i + j + 1.0;
}

__host__ __device__ void init_B(matrix_t *mat, int i, int j)
{
	mat->contents[calc_matrix_offset(mat, i, j)] = i + 4.0 * j + 2.0;
}


__global__ void setupA(matrix_t *mat) {
	const int m = mat->m;
	const int n = mat->n;
	//initial location for the thread
	int dev_m = blockIdx.x * blockDim.x + threadIdx.x;
	int dev_n = blockIdx.y * blockDim.y + threadIdx.y;
	//thread's done when its index goes off the bottom 
	while (dev_n < n){
		// if we're within the range of the array
		// do the init of A
		if ((dev_m < m) && (dev_n < n)) {
			//init_A(mat, dev_m, dev_n);
			mat->contents[calc_matrix_offset(mat, dev_m, dev_n)] = 2.0 * dev_m + dev_n + 1.0;
		}
		// shift the index forward
		dev_m += gridDim.x*blockDim.x;
		// if the new index is outside of the range
		// reset to the start with a mod
		// increment the n dimension i.e. next row
		if (!(dev_m < m)) {
			dev_m = dev_m % m;
			dev_n = dev_n + gridDim.y*blockDim.y;
		}
	}
}

__global__ void setupB(matrix_t *mat) {
	const int m = mat->m;
	const int n = mat->n;
	int dev_m = blockIdx.x * blockDim.x + threadIdx.x;
	int dev_n = blockIdx.y * blockDim.y + threadIdx.y;
	while (dev_n < n) {
		if ((dev_m < m) && (dev_n < n)) {
			//init_B(mat, dev_m, dev_n);
			mat->contents[calc_matrix_offset(mat, dev_m, dev_n)] = dev_m + 4.0 * dev_n + 2.0;
		}
		dev_m += gridDim.x*blockDim.x;
		if (!(dev_m < m)) {
			dev_m = dev_m % m;
			dev_n = dev_n + gridDim.y*blockDim.y;
		}
	}
}

__global__ void add(matrix_t *a, matrix_t *b, matrix_t *c)
{
	const int m = a->m;
	const int n = a->n;
	int dev_m = blockIdx.x * blockDim.x + threadIdx.x; 
	int dev_n = blockIdx.y * blockDim.y + threadIdx.y;
	float item_a = 0;
	float item_b = 0;
	float item_c = 0;
	while (dev_n < n) {
		if ((dev_m < m) && (dev_n < n)) {
			//float temp = get_matrix_item(a, dev_m, dev_n) + get_matrix_item(b, dev_m, dev_n);
			//set_matrix_item(c, dev_m, dev_n, temp);
			item_a = a->contents[calc_matrix_offset(a, dev_m, dev_n)];
			item_b = b->contents[calc_matrix_offset(b, dev_m, dev_n)];
			item_c = item_a + item_b;
			c->contents[calc_matrix_offset(c, dev_m, dev_n)] = item_c;
		}
		dev_m += gridDim.x*blockDim.x;
		if (!(dev_m < m)) {
			dev_m = dev_m % m;
			dev_n = dev_n + gridDim.y*blockDim.y;
		}
	}
}